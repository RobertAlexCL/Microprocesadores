#include "hip/hip_runtime.h"
/**
 * ------------
 * TFNilaka.cu
 * ------------
 * AUTH.	Roberto Castillo
 * DATE		2020-11-24
 * ------------
 * Universidad del Valle de Guatemala
 * CC3056 - Programación de Microprocesadores
 * 
 */

#include <stdio.h>
#include <iostream>


#define N 1024
#define bsize 32
__global__ void nilakantha(double* arreglo)
{
	int hilo = blockIdx.x * blockDim.x + threadIdx.x;
	double k = (hilo+1) * 2;
	if (hilo < N){
		if (hilo % 2 == 0){
			arreglo[hilo] = 4/(k*(k+1)*(k+2));
		}
		else{
			
			arreglo[hilo] = -4/(k*(k+1)*(k+2));
		}
		
	}
}

int main()
{
	double *st, *d_st;
	int size = sizeof(double);

	st = (double*)malloc(size*N);
	hipMalloc((void **) &d_st, size*N);

	nilakantha<<<N/bsize, bsize>>>(d_st);
	
	hipMemcpy(st, d_st, size*N, hipMemcpyDeviceToHost);

	double pi = 3;
	int h;
	for(h = 0; h<N; h++;)
		pi += st[h];


	printf("Pi: %.10f\n", pi);

	free(st);
	hipFree(d_st);
	return 0;
}